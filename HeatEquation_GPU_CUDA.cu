#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <time.h>
#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <math.h>
#define _USE_MATH_DEFINES

#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "lodepng.h"

typedef uint8_t u8;
typedef uint32_t u32;
typedef float f32;
typedef double f64;
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

f64 measureTime() {
    struct timespec now;
    u32 t = timespec_get(&now, TIME_UTC);
    return now.tv_sec + now.tv_nsec * 1e-9;
}

u32 __host__ __device__ getIndex(u32 i, u32 j, u32 width) {
    return i * width + j;
}

void __global__ heatFlowGlobal(f32* T_old, f32* T_new, u32 size_xy, f32 dx2, f32 eta, f32 dt) {
    u32 i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > 0 && i < size_xy - 1) {
        u32 j = threadIdx.y + blockIdx.y * blockDim.y;
        if (j > 0 && j < size_xy - 1) {
            u32 k = i * size_xy + j;
            u32 kim1 = (i - 1) * size_xy + j;
            u32 kip1 = (i + 1) * size_xy + j;
            u32 kjm1 = i * size_xy + j - 1;
            u32 kjp1 = i * size_xy + j + 1;
            T_new[k] = T_old[k] + dt * eta * (T_old[kim1] + T_old[kip1] + T_old[kjm1] + T_old[kjp1] - 4.0 * T_old[k])/dx2;
        }
    }
}

__global__ void heatFlowShared(f32* T_old, f32* T_new, u32 size_xy, f32 dx2, f32 eta, f32 dt) {
    u32 i = threadIdx.x + blockIdx.x * blockDim.x;
    u32 j = threadIdx.y + blockIdx.y * blockDim.y;
    u32 k = i * size_xy + j;

    __shared__ f32 s_T_old[BLOCK_SIZE_X + 2][BLOCK_SIZE_Y + 2];

    if (i < size_xy && j < size_xy) {
        s_T_old[threadIdx.x + 1][threadIdx.y + 1] = T_old[k];
        if (threadIdx.x == 0 && i > 0) {
            s_T_old[0][threadIdx.y + 1] = T_old[k - size_xy];
        }
        if (threadIdx.x == BLOCK_SIZE_X - 1 && i < size_xy - 1) {
            s_T_old[BLOCK_SIZE_X + 1][threadIdx.y + 1] = T_old[k + size_xy];
        }
        if (threadIdx.y == 0 && j > 0) {
            s_T_old[threadIdx.x + 1][0] = T_old[k - 1];
        }
        if (threadIdx.y == BLOCK_SIZE_Y - 1 && j < size_xy - 1) {
            s_T_old[threadIdx.x + 1][BLOCK_SIZE_Y + 1] = T_old[k + 1];
        }
    }

    __syncthreads();

    if (i > 0 && i < size_xy - 1 && j > 0 && j < size_xy - 1) {
        T_new[k] = s_T_old[threadIdx.x + 1][threadIdx.y + 1] +
            dt * eta *
            (s_T_old[threadIdx.x][threadIdx.y + 1] +
                s_T_old[threadIdx.x + 2][threadIdx.y + 1] +
                s_T_old[threadIdx.x + 1][threadIdx.y] +
                s_T_old[threadIdx.x + 1][threadIdx.y + 2] -
                4.0 * s_T_old[threadIdx.x + 1][threadIdx.y + 1]) /
            dx2;
    }
}

u32 main() {
    f64 start1 = measureTime();

    u32 size_xy = 4096;
    u32 n_steps = 10000;
    f32 eta = 1.0;
    f32 dx = 1.0 / size_xy;
    f32 dx2 = dx * dx;
    f32 dt = (dx2 * dx2) / (4.0 * eta * dx2);
    f64 pi = acos(-1);

    f32* T_old = (f32*)malloc(sizeof(*T_old) * size_xy * size_xy);
    for (u32 i = 0; i < size_xy * size_xy; i++) {
        T_old[i] = 0.0;
    }

    f32* T_old_d;
    f32* T_new_d;

    hipMalloc((void**)(&T_old_d), size_xy * size_xy * sizeof(f32));
    hipMalloc((void**)(&T_new_d), size_xy * size_xy * sizeof(f32));

    u8* image = (u8*)malloc(sizeof(*image) * size_xy * size_xy * 3);

    dim3 numBlocks(size_xy / BLOCK_SIZE_X + 1, size_xy / BLOCK_SIZE_Y + 1);
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    for (u32 i = 0; i < size_xy; i++) {
        u32 k_left = i * size_xy;
        u32 k_right = i * size_xy + size_xy - 1;
        u32 k_top = i;
        u32 k_bottom = (size_xy - 1) * size_xy + i;

        T_old[k_left] = 255.0 * cos(i * pi / float(size_xy)) * cos(i * pi / float(size_xy));
        T_old[k_right] = 255.0 * sin(i * pi / float(size_xy)) * sin(i * pi / float(size_xy));
        T_old[k_top] = 255.0 - 255.0 * i / float(size_xy);
        T_old[k_bottom] = 255.0 - 255.0 * i / float(size_xy);
    }

    hipMemcpy(T_old_d, T_old, size_xy * size_xy * sizeof(f32), hipMemcpyHostToDevice);
    hipMemcpy(T_new_d, T_old, size_xy * size_xy * sizeof(f32), hipMemcpyHostToDevice);

    f64 end1 = measureTime();

    printf("Grid Creation and Initialization Time: %.9lfs\n", end1 - start1);

    if (image != NULL) {
        f64 start2 = measureTime();
        for (u32 n = 0; n < n_steps; n++) {
            heatFlowGlobal <<< numBlocks, threadsPerBlock >>> (T_old_d, T_new_d, size_xy, dx2, eta, dt);
            hipDeviceSynchronize();

            if (n % 1000 == 0) {
                hipMemcpy(T_old, T_old_d, size_xy * size_xy * sizeof(f32), hipMemcpyDeviceToHost);
                hipError_t errorCode = hipGetLastError();
                if (errorCode != hipSuccess) {
                    printf("Cuda Error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                    exit(0);
                }
                char filename[100] = "";
                sprintf(filename, "pics/heat_%05d.png", n);
                for (u32 i = 0; i < size_xy * size_xy; i++) {
                    image[i * 3] = (u8)T_old[i];
                    image[i * 3 + 1] = 0;
                    image[i * 3 + 2] = 0;
                }
                u32 error = lodepng_encode24_file(filename, image, size_xy, size_xy);
                if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
            }
            
            f32* temp = T_old_d;
            T_old_d = T_new_d;
            T_new_d = temp;
        }

        free(T_old);

        hipFree(T_old_d);
        hipFree(T_new_d);

        f64 end2 = measureTime();
        printf("Grid Creation and Initialization Time: %.9lfs\n", end2 - start2);

    }
    return 0;
}
